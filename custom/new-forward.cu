#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <hip/hip_fp16.h>
#include "gpu-new-forward.h"

#define cuda_check(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            std::cout << "Failed to run stmt " << #stmt << std::endl;                       \
            std::cout << "Got CUDA error ...  " << hipGetErrorString(err) << std::endl;    \
            exit(-1);                                                        \
        }                                                                     \
    } while(0)

#define y4d(y, i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(x, i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(k, i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
#define for_in(v, max) for (int v = 0; v < max; v++)

__constant__ float constant_k[16 * 4 * 7 * 7];

#define TILE_WIDTH 16
__global__ void conv_forward_kernel(float *y, float *x, const int B, const int M, const int C, const int H, const int W, const int K)
{
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    const int w_grid = (int) ceil(((float) W_out) / TILE_WIDTH);

	// Recover the blockidx in x and y dim using Z
	const int H_bi = blockIdx.z / w_grid;
	const int W_bi = blockIdx.z % w_grid;
	
	// h and w refers to both the output and input index(some of the threads will be turned off during output)
    const int h = H_bi * TILE_WIDTH + threadIdx.y;
    const int w = W_bi * TILE_WIDTH + threadIdx.x;
    const int b = blockIdx.x;
    const int m = blockIdx.y;

    if (h < H_out && w < W_out) {
        float acc = 0;
        for_in(c, C) {
            for_in(p, K) {
                for_in(q, K) {
                    acc += x4d(x, b, c, h + p, w + q) * k4d(constant_k, m, c, p, q);
                }
            }
        }
        y4d(y, b, m, h, w) = acc;
    }
}

__host__ void GPUInterface::conv_forward_gpu(float *host_y, const float *host_x, const float *host_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Declare relevant device pointers
    float *dev_x, *dev_y, *dev_k;
    float *dev_k_16;
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    // Allocate memory and copy over the relevant data structures to the GPU
    size_t x_len = B * C * H * W;
    size_t y_size = B * M * H_out * W_out * sizeof(float);
    size_t k_len = M * C * K * K;
    cuda_check(hipMalloc(&dev_x, x_len * sizeof(float)));
    cuda_check(hipMalloc(&dev_y, y_size));
    cuda_check(hipMalloc(&dev_k, k_len * sizeof(float)));
    cuda_check(hipMalloc(&dev_k_16, k_len * sizeof(float)));
    cuda_check(hipMemcpyToSymbol(HIP_SYMBOL(constant_k), host_k, k_len * sizeof(float)));

    const int H_grid = ceil((float) H_out / TILE_WIDTH);
    const int W_grid = ceil((float) W_out / TILE_WIDTH);
    const int Z = H_grid * W_grid;
    dim3 dim_grid(B, M, Z);
    dim3 dim_block(TILE_WIDTH, TILE_WIDTH, 1);

    cuda_check(hipMemcpy(dev_x, host_x, x_len * sizeof(float), hipMemcpyHostToDevice));
    conv_forward_kernel<<<dim_grid, dim_block>>>(dev_y, dev_x, B, M, C, H, W, K);
    cuda_check(hipMemcpy(host_y, dev_y, y_size, hipMemcpyDeviceToHost));

    // Free device memory
    cuda_check(hipFree(dev_x));
    cuda_check(hipFree(dev_k_16));
    cuda_check(hipFree(dev_y));
    cuda_check(hipFree(dev_k));
}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
